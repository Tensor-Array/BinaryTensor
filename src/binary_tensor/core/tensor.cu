#include "hip/hip_runtime.h"
/*
Copyright 2024 TensorArray-Creators

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <cstring>
#include <cstdio>
#include <cassert>
#include <exception>
#include <hip/hip_fp8.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#ifndef TENSOR_CONTENT
#define TENSOR_CONTENT
#include "tensor.hh"
#undef TENSOR_CONTENT
#endif // !TENSOR_CONTENT

#define USING_DATA_TYPE (uint1_t_x8)

#define LOOP(seq) END(A seq)
#define BODY(x) ADD_CODE(x)
#define A(x) BODY(x) B
#define B(x) BODY(x) A
#define A_END
#define B_END
#define END(...) END_(__VA_ARGS__)
#define END_(...) __VA_ARGS__##_END

namespace binary_tensor
{
    namespace value
    {
		using namespace devices;

		template <typename T>
		__global__ void set_values(T value_arr[], T value, unsigned int c_size)
		{
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			if (thread_x < c_size)
				value_arr[thread_x] = value;
		}

		__global__ void set_values_random(float value_arr[], unsigned long long seed, unsigned int max_size)
		{
			hiprandState thisState;
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			hiprand_init(seed, thread_x, HIPRAND_RNG_TEST, &thisState);
			if (thread_x < max_size)
				value_arr[thread_x] = hiprand_uniform(&thisState);
		}

		template <typename T>
		__global__ void sum_2_arr(T c[], const T a[], const T b[], unsigned int c_size)
		{
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			if (thread_x < c_size)
				c[thread_x] = a[thread_x] + b[thread_x];
		}

		template <typename T>
		__global__ void mul_2_arr(T c[], const T a[], const T b[], unsigned int c_size)
		{
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			if (thread_x < c_size)
				c[thread_x] = a[thread_x] * b[thread_x];
		}

		template <typename T>
		__global__ void div_2_arr(T c[], const T a[], const T b[], unsigned int c_size)
		{
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			if (thread_x < c_size)
				c[thread_x] = a[thread_x] / b[thread_x];
		}

		template <typename T>
		__global__ void array_condition(T out_value[], unsigned int c_size, const bool bool_value[], const T true_value[], const T false_value[])
		{
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			if (thread_x < c_size)
				out_value[thread_x] = bool_value[thread_x] ? true_value[thread_x] : false_value[thread_x];
		}

		__global__ void kernel_transpose(void* output, const void* input, unsigned int c_size, unsigned int dim_1_size, unsigned int dim_2_size , size_t child_size)
		{
			unsigned int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
			unsigned int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
			unsigned int thread_z = blockIdx.z * blockDim.z + threadIdx.z;
			if (thread_x < c_size && thread_y < dim_1_size && thread_z < dim_2_size)
				std::memcpy
				(
					reinterpret_cast<void*>(reinterpret_cast<size_t>(output) +
						thread_x * dim_2_size * dim_1_size * child_size +
						thread_z * dim_1_size * child_size +
						thread_y * child_size),
					reinterpret_cast<const void*>(reinterpret_cast<size_t>(input) +
						thread_x * dim_1_size * dim_2_size * child_size +
						thread_y * dim_2_size * child_size +
						thread_z * child_size),
					child_size
				);
		}

		bool equal_dim_size(const TensorBase& a, const TensorBase& b);

		Tensor derive_transpose(const Tensor& a, const Tensor&, bool is_derive, const DataBuffer&)
		{
			const std::vector<unsigned int> shape_a = a.get_buffer().shape();
			assert(shape_a.size() == 4);
			std::vector<std::pair<Tensor, Derivation>> temp;
			if (is_derive)
			{
				temp.push_back(std::make_pair(a, Derivation(Tensor(), derive_transpose)));
			}
			hipError_t cudaStat;
			devices::Device this_cuda{ devices::CUDA };
			cudaStat = hipGetDevice(&this_cuda.index);
			hipDeviceProp_t cu_dev_prop;
			cudaStat = hipGetDeviceProperties(&cu_dev_prop, this_cuda.index);
			TensorBase base_a = a.get_buffer().change_device(this_cuda);
			void* c_ptr;
			cudaStat = hipMalloc(&c_ptr, base_a.data_size());
			cudaStat = hipMemset(c_ptr, 0, a.get_buffer().data_size());
			dim3 block_dim(1, 32, 32);
			dim3 grid_dim(
				shape_a.begin()[0] / block_dim.x + (shape_a.begin()[0] % block_dim.x ? 1U : 0U),
				shape_a.begin()[1] / block_dim.y + (shape_a.begin()[1] % block_dim.y ? 1U : 0U),
				shape_a.begin()[2] / block_dim.z + (shape_a.begin()[2] % block_dim.z ? 1U : 0U)
			);
			kernel_transpose<<<grid_dim, block_dim>>>(c_ptr, base_a.data(), shape_a.begin()[0], shape_a.begin()[1], shape_a.begin()[2], shape_a.begin()[3] * get_sizeof_type(a.get_buffer().type()));
			cudaStat = hipDeviceSynchronize();
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess)
			{
				printf("CUDA error: %s\n", hipGetErrorString(cudaStat));
			}
			TensorBase value_buf({ shape_a.begin()[0], shape_a.begin()[2], shape_a.begin()[1], shape_a.end()[-1]}, c_ptr, this_cuda);
			cudaStat = hipFree(c_ptr);
			return Tensor(std::move(value_buf), std::move(temp));
		}
		
		Tensor values(const std::initializer_list<unsigned int>& list_dim, uint1_t_x8 value)
		{
			hipError_t cudaStatus;
			uint1_t_x8* dev_ptr;
			unsigned int total_size = 1;
			for (unsigned int i: list_dim)
				total_size *= i;
			devices::Device this_cuda{ devices::CUDA };
			cudaStatus = hipGetDevice(&this_cuda.index);
			hipDeviceProp_t cu_dev_prop;
			cudaStatus = hipGetDeviceProperties(&cu_dev_prop, this_cuda.index);
			cudaStatus = hipMalloc(&dev_ptr, total_size * sizeof(float));
			dim3 block_dim(cu_dev_prop.maxThreadsDim[0]);
			dim3 grid_dim(total_size / block_dim.x + (total_size % block_dim.x ? 1U : 0U));
			set_values << <grid_dim, block_dim>> > (dev_ptr, value, total_size);
			cudaStatus = hipDeviceSynchronize();
			assert((cudaStatus = hipGetLastError()) == hipSuccess);
			if (cudaStatus != hipSuccess);
			TensorBase other_buf(list_dim, dev_ptr, this_cuda);
			cudaStatus = hipFree(dev_ptr);
			return Tensor(std::move(other_buf));
		}

		Tensor tensor_rand(const std::initializer_list<unsigned int>& list_dim, unsigned int seed)
		{
			hipError_t cudaStatus;
			float* dev_ptr;
			unsigned int total_size = 1;
			for (unsigned int i: list_dim)
				total_size *= i;
			devices::Device this_cuda{ devices::CUDA };
			cudaStatus = hipGetDevice(&this_cuda.index);
			hipDeviceProp_t cu_dev_prop;
			cudaStatus = hipGetDeviceProperties(&cu_dev_prop, this_cuda.index);
			cudaStatus = hipMalloc(&dev_ptr, total_size * sizeof(float));
			dim3 block_dim(cu_dev_prop.maxThreadsDim[0]);
			dim3 grid_dim(total_size / block_dim.x + (total_size % block_dim.x ? 1U : 0U));
			set_values_random << <grid_dim, block_dim>> > (dev_ptr, seed, total_size);
			cudaStatus = hipDeviceSynchronize();
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
			{
				printf("CUDA error: %s\n", hipGetErrorString(cudaStatus));
			}
			TensorBase other_buf(list_dim, dev_ptr, this_cuda);
			cudaStatus = hipFree(dev_ptr);
			return other_buf;
		}

		Tensor multiply(const Tensor& a, const Tensor& b, bool is_derive, const DataBuffer&)
		{
			assert(equal_dim_size(a.get_buffer(), b.get_buffer()));
			std::vector<std::pair<Tensor, Derivation>> temp;
			if (is_derive)
			{
				temp.push_back(std::make_pair(a, Derivation(b.clone(), multiply)));
				temp.push_back(std::make_pair(b, Derivation(a.clone(), multiply)));
			}
			hipError_t cuda_status;
			TensorBase other_buf;
			void* c_ptr;
			devices::Device this_cuda{ devices::CUDA };
			cuda_status = hipGetDevice(&this_cuda.index);
			hipDeviceProp_t cu_dev_prop;
			cuda_status = hipGetDeviceProperties(&cu_dev_prop, this_cuda.index);
			TensorBase base_a = a.get_buffer().change_device(this_cuda);
			TensorBase base_b = b.get_buffer().change_device(this_cuda);
			std::size_t c_size = std::max
			(
				a.get_buffer().data_size() / get_sizeof_type(a.get_buffer().type()),
				b.get_buffer().data_size() / get_sizeof_type(b.get_buffer().type())
			);
			cuda_status = hipMalloc(&c_ptr, std::max(a.get_buffer().data_size(), b.get_buffer().data_size()));
			dim3 block_dim(cu_dev_prop.maxThreadsDim[0]);
			dim3 grid_dim(c_size / block_dim.x + (c_size % block_dim.x ? 1U : 0U));
#define ADD_CODE(TYPE) \
if(a.get_buffer().type() == typeid(TYPE) && b.get_buffer().type() == typeid(TYPE)) \
{ \
mul_2_arr<<<grid_dim, block_dim>>>(static_cast<TYPE*>(c_ptr), static_cast<const TYPE*>(base_a.data()), static_cast<const TYPE*>(base_b.data()), c_size); \
cuda_status = hipDeviceSynchronize(); \
other_buf = TensorBase(a.get_buffer().shape(), c_ptr, this_cuda); \
}
			LOOP(USING_DATA_TYPE);
#undef ADD_CODE
			cuda_status = hipFree(c_ptr);
			return Tensor(std::move(other_buf), std::move(temp));
		}

		Tensor condition(const Tensor& bool_value, const Tensor& true_value, const Tensor& false_value, bool is_derive)
		{
			assert(
				equal_dim_size(bool_value.get_buffer(), true_value.get_buffer()) &&
				equal_dim_size(bool_value.get_buffer(), false_value.get_buffer()) &&
				bool_value.get_buffer().type() == typeid(bool) &&
				true_value.get_buffer().type() == false_value.get_buffer().type()
			);
			std::vector<std::pair<Tensor, Derivation>> temp;
			if (is_derive)
			{
				Tensor value_ones = ones(bool_value.get_buffer().shape());
				Tensor value_zeros = zeros(bool_value.get_buffer().shape());
				temp.push_back(std::make_pair(true_value, Derivation(condition(bool_value, value_ones, value_zeros, false), multiply)));
				temp.push_back(std::make_pair(false_value, Derivation(condition(bool_value, value_zeros, value_ones, false), multiply)));
			}
			hipError_t cuda_status;
			TensorBase other_buf;
			void* ptr_out;
			devices::Device this_cuda{ devices::CUDA };
			cuda_status = hipGetDevice(&this_cuda.index);
			hipDeviceProp_t cu_dev_prop;
			cuda_status = hipGetDeviceProperties(&cu_dev_prop, this_cuda.index);
			TensorBase base_bool_value = bool_value.get_buffer().change_device(this_cuda);
			TensorBase base_true_value = true_value.get_buffer().change_device(this_cuda);
			TensorBase base_false_value = false_value.get_buffer().change_device(this_cuda);
			std::size_t c_size = base_bool_value.data_size();
			dim3 block_dim(cu_dev_prop.maxThreadsDim[0]);
			dim3 grid_dim(c_size / block_dim.x + (c_size % block_dim.x ? 1U : 0U));
#define ADD_CODE(TYPE) \
if(true_value.get_buffer().type() == typeid(TYPE) && false_value.get_buffer().type() == typeid(TYPE)) \
{ \
cuda_status = hipMalloc(&ptr_out, c_size * sizeof(TYPE));\
array_condition<<<grid_dim, block_dim>>>(static_cast<TYPE*>(ptr_out), c_size, static_cast<const bool*>(base_bool_value.data()), static_cast<const TYPE*>(base_true_value.data()), static_cast<const TYPE*>(base_false_value.data())); \
cuda_status = hipDeviceSynchronize(); \
other_buf = TensorBase(bool_value.get_buffer().shape(), ptr_out, this_cuda); \
}
			LOOP(USING_DATA_TYPE);
#undef ADD_CODE
			cuda_status = hipFree(ptr_out);
			return Tensor(std::move(other_buf), std::move(temp));
		}

		Tensor add(const Tensor& a, const Tensor& b, bool is_derive)
		{
			assert(equal_dim_size(a.get_buffer(), b.get_buffer()));
			std::vector<std::pair<Tensor, Derivation>> temp;
			if (is_derive)
			{
				temp.push_back(std::make_pair(a, Derivation(ones(a.get_buffer().shape()), multiply)));
				temp.push_back(std::make_pair(b, Derivation(ones(b.get_buffer().shape()), multiply)));
			}
			hipError_t cuda_status;
			TensorBase other_buf;
			void* c_ptr;
			devices::Device this_cuda{ devices::CUDA };
			cuda_status = hipGetDevice(&this_cuda.index);
			hipDeviceProp_t cu_dev_prop;
			cuda_status = hipGetDeviceProperties(&cu_dev_prop, this_cuda.index);
			TensorBase base_a = a.get_buffer().change_device(this_cuda);
			TensorBase base_b = b.get_buffer().change_device(this_cuda);
			std::size_t c_size = std::max
			(
				a.get_buffer().data_size() / get_sizeof_type(a.get_buffer().type()),
				b.get_buffer().data_size() / get_sizeof_type(b.get_buffer().type())
			);
			cuda_status = hipMalloc(&c_ptr, std::max(a.get_buffer().data_size(), b.get_buffer().data_size()));
			dim3 block_dim(cu_dev_prop.maxThreadsDim[0]);
			dim3 grid_dim(c_size / block_dim.x + (c_size % block_dim.x ? 1U : 0U));
#define ADD_CODE(TYPE) \
if(a.get_buffer().type() == typeid(TYPE) && b.get_buffer().type() == typeid(TYPE)) \
{ \
sum_2_arr<<<grid_dim, block_dim>>>(static_cast<TYPE*>(c_ptr), static_cast<const TYPE*>(base_a.data()), static_cast<const TYPE*>(base_b.data()), c_size); \
cuda_status = hipDeviceSynchronize(); \
other_buf = TensorBase(a.get_buffer().shape(), c_ptr, this_cuda); \
}
			LOOP(USING_DATA_TYPE);
#undef ADD_CODE
			cuda_status = hipFree(c_ptr);
			return Tensor(std::move(other_buf), std::move(temp));
		}
		
    }
}

#undef LOOP
#undef BODY
#undef A
#undef B
#undef A_END
#undef B_END
#undef END
#undef END_

#undef USING_DATA_TYPE
